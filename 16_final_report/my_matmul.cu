#include "hip/hip_runtime.h"
#include <mpi.h>
#include <cstdio>
#include <cmath>
#include <cstdlib>
#include <vector>
#include <chrono>
using namespace std;

__global__ void matmul(float *A, float *B, float *C, int N, int offset, int size) {
  int i = blockIdx.y;
  int j = threadIdx.x + blockDim.x * blockIdx.x;
  float sum = 0.0f;
  extern __shared__ float A_s[];
  for (int ks=0; ks<N; ks+=blockDim.x) {
    __syncthreads();
    A_s[threadIdx.x] = A[N*i+ks+threadIdx.x];
    __syncthreads();
    for (int k=ks; k<ks+blockDim.x; k++) {
      sum += A_s[k-ks] * B[N/size*k+j];
    }
  }
  C[N*i+j+offset] = sum;
}

int main(int argc, char** argv) {
  int size, rank;
  MPI_Init(&argc, &argv);
  MPI_Comm_size(MPI_COMM_WORLD, &size);
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);

  const int N = 2048;
  const int M = 256;
  vector<float> A(N*N);
  vector<float> B(N*N);
  vector<float> C(N*N, 0);
  float *subA, *subB, *subC;
  vector<float> recv(N*N/size);
  int size_sub = (N*N/size) * sizeof(float);
  hipMallocManaged(&subA, size_sub);
  hipMallocManaged(&subB, size_sub);
  hipMallocManaged(&subC, size_sub);
  dim3 grid(N/size/M, N/size);
  for (int i=0; i<N; i++) {
    for (int j=0; j<N; j++) {
      A[N*i+j] = drand48();
      B[N*i+j] = drand48();
    }
    for (int j=0; j<N/size; j++) {
      subC[N*j+i] = 0;
    }
  }
  int offset = N/size*rank;
  for (int i=0; i<N/size; i++)
    for (int j=0; j<N; j++)
      subA[N*i+j] = A[N*(i+offset)+j];
  for (int i=0; i<N; i++)
    for (int j=0; j<N/size; j++)
      subB[N/size*i+j] = B[N*i+j+offset];
  int recv_from = (rank + 1) % size;
  int send_to = (rank - 1 + size) % size;

  double comp_time = 0, comm_time = 0;
  for(int irank=0; irank<size; irank++) {
    auto tic = chrono::steady_clock::now();
    offset = N/size*((rank+irank) % size);


    // for (int i=0; i<N/size; i++)
    //   for (int j=0; j<N/size; j++)
    //     for (int k=0; k<N; k++)
    //       subC[N*i+j+offset] += subA[N*i+k] * subB[N/size*k+j];
    matmul<<<grid,M,M*sizeof(float)>>>(subA, subB, subC, N, offset, size);
    hipDeviceSynchronize();

    
    auto toc = chrono::steady_clock::now();
    comp_time += chrono::duration<double>(toc - tic).count();
    MPI_Request request[2];
    MPI_Isend(&subB[0], N*N/size, MPI_FLOAT, send_to, 0, MPI_COMM_WORLD, &request[0]);
    MPI_Irecv(&recv[0], N*N/size, MPI_FLOAT, recv_from, 0, MPI_COMM_WORLD, &request[1]);
    MPI_Waitall(2, request, MPI_STATUS_IGNORE);
    for (int i=0; i<N*N/size; i++)
      subB[i] = recv[i];
    tic = chrono::steady_clock::now();
    comm_time += chrono::duration<double>(tic - toc).count();
  }
  MPI_Allgather(&subC[0], N*N/size, MPI_FLOAT, &C[0], N*N/size, MPI_FLOAT, MPI_COMM_WORLD);
  for (int i=0; i<N; i++)
    for (int j=0; j<N; j++)
      for (int k=0; k<N; k++)
        C[N*i+j] -= A[N*i+k] * B[N*k+j];
  double err = 0;
  for (int i=0; i<N; i++)
    for (int j=0; j<N; j++)
      err += fabs(C[N*i+j]);
  if(rank==0) {
    double time = comp_time+comm_time;
    printf("N    : %d\n",N);
    printf("comp : %lf s\n", comp_time);
    printf("comm : %lf s\n", comm_time);
    printf("total: %lf s (%lf GFlops)\n",time,2.*N*N*N/time/1e9);
    printf("error: %lf\n",err/N/N);
  }
  MPI_Finalize();
  hipFree(subA);
  hipFree(subB);
  hipFree(subC);
}
